
#include "hip/hip_runtime.h"
#include ""
#include "highPerfTimer.h"
#include <stdio.h>
#include <fstream>
#include <stdlib.h>
#include <iostream>
using namespace std;


int main() {
	//create the thing with the stuff for the position
	int giga = 1 << 30;


	//add timer
	HighPrecisionTime hpt;

	//array of char, this will be a buffer
	char *buffer = nullptr; // (char*)malloc(giga);
	buffer = new char[giga]();
	//create bitmap
	char * bitmap = nullptr;
	bitmap = new char[giga / 8]();


	//int * bitmap = (int *)malloc(giga / 8);

	//open file
	ifstream enWiki("C:/Users/educ/Documents/enwiki-latest-abstract.xml");
	if (enWiki.fail()) {
		cout << "Cannot open file " << endl;
		return 1;
	}

	//start timer for loading the file
	float startTime = hpt.TimeSinceLastCall();
	//file into buffer
	enWiki.read(buffer, giga);


	//end and print timer
	cout << "Loading file took: " << hpt.TimeSinceLastCall() - startTime << endl;
	//close
	free(buffer);
	enWiki.close();


	system("pause");
	return 0;
}