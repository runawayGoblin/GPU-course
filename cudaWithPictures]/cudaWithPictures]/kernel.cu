#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include<iostream>


using namespace cv;
using namespace std;

void CpuThreshold(int threshold, int width, int height, unsigned char* data);
hipError_t mallocGPU(unsigned char** orig, unsigned char** modif, int width, int height);
void cleanGPU(unsigned char* orig, unsigned char*);
hipError_t copyToGPU(unsigned char* gpuData, unsigned char* cpuData, int width, int height);
hipError_t copyToCPU(unsigned char* gpuData, unsigned char* cpuData, int width, int height);
void onTrack(int thr, void* pt);
__global__ void gpuThresholdKernel(unsigned char* gpuOrig, unsigned char* gpuModif, int sizeArr, int threshold) {
	//loop, compare threshold, change vals
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < sizeArr) {
		if (gpuOrig[i] > threshold) {
			gpuModif[i] = 255;
		}
		else {
			gpuModif[i] = 0;
		}
	}



}
//global vars
unsigned char * gpuOriginalImage = nullptr;
unsigned char * gpuModifiedImage = nullptr;
Mat cpuOriginalImage;
int height;
int width;
int thresholdSlider = 195;

int main(int argc, char** argv) {
	
	if (argc != 2) {
		cout << "usage: display_image ImageToLoadAndDisplay" << endl;
	}

	/*Mat cpuOriginalImage;*/
	cpuOriginalImage = imread(argv[1], CV_LOAD_IMAGE_COLOR);
	cvtColor(cpuOriginalImage, cpuOriginalImage, COLOR_RGB2GRAY);

	cout << "Number of channels: " << cpuOriginalImage.channels() << endl;
	if (!cpuOriginalImage.data) {
		cout << "could not open or find the image" << std::endl;
	}
	height = cpuOriginalImage.rows;
	width = cpuOriginalImage.cols;
	//int threshold = 14;

	//declare vairables to put the data into
	//Mat.data returns a pointer to an unsighned character array
	/*unsigned char * gpuOriginalImage;
	unsigned char * gpuModifiedImage=0;
	*/	
	CpuThreshold(thresholdSlider, width, height, cpuOriginalImage.data);
	cout << "CpuThreshold was a success" << endl << endl;
	try {
		//create space on the gpu to hold the modified and the image data
		hipError_t gpuStatus = mallocGPU(&gpuOriginalImage, &gpuModifiedImage, width, height);
		if (gpuStatus != hipSuccess) {
			throw("Malloc GPU Failed");
		}
		cout << "gpuMalloc was a success" << endl << endl;
		
		//copy the cpu image data to the gpu
		gpuStatus = copyToGPU(gpuOriginalImage, cpuOriginalImage.data, width, height);
		if (gpuStatus != hipSuccess) {
			throw("Copy to GPU Failed");
		}
		cout << "hipMemcpy cpuImg to gpuOrig Worked" << endl << endl;

		//////update the image with the threshold
		//int numBlocks = (1023 + width * height) / 1024;
		//gpuThresholdKernel <<<numBlocks, 1024 >>> (gpuOriginalImage, gpuModifiedImage,(width * height), thresholdSlider);
		////time(??)
		//gpuStatus = hipGetLastError();
		//if (gpuStatus != hipSuccess) {
		//	throw("Kernel Failed");
		//}
		////cout << "Kernel Worked" << endl << endl;


		////copy back to cpu
		//gpuStatus = copyToCPU(gpuOriginalImage, cpuOriginalImage.data, width, height);
		//if (gpuStatus != hipSuccess) {
		//	throw("Copy to GPU Failed");
		//}
		//cout << "cudaMemcpu to Cpu Worked" << endl << endl;


	}
	catch (char* errMsg) {
		cout << "Error: " << errMsg << endl;
		cleanGPU(gpuOriginalImage, gpuModifiedImage);
	}
	
	namedWindow("Display Window", WINDOW_NORMAL);
	//imshow("Display Window", cpuOriginalImage);
	createTrackbar("Slider", "Display Window", &thresholdSlider, 255, onTrack);
	onTrack(thresholdSlider, 0);

	waitKey(0);
	cleanGPU(gpuOriginalImage, gpuModifiedImage);
	return 0;
}
void CpuThreshold(int threshold, int width, int height, unsigned char* data) {

	unsigned char* endArr = data + (width * height);
	//loop through data 
	for (unsigned char * index = data; index < endArr; index++) {
		//if data at index is greater than threshold index=255
		if (*index > threshold) {
			*index = 255;
		}
		else {//otherwise index = 0
			*index = 0;
		}
		
	}

}
void onTrack(int, void* ) {
	hipError_t gpuStatus = hipSuccess;
	
	int numBlocks = (1023 + width * height) / 1024;
	gpuThresholdKernel << <numBlocks, 1024 >> > (gpuOriginalImage, gpuModifiedImage, (width * height), thresholdSlider);
	
	gpuStatus = hipGetLastError();
	if (gpuStatus != hipSuccess) {
		cout << "nopeA" << endl;
	}
	
	cout << "threshold: " << thresholdSlider << endl;

	gpuStatus = hipDeviceSynchronize();
	if (gpuStatus != hipSuccess){
		cout << "ooooopppppss" << endl;
	}

	//copy back to cpu
	gpuStatus = copyToCPU(gpuModifiedImage, cpuOriginalImage.data, width, height);
	if (gpuStatus != hipSuccess) {
		cout << "gpu copy failed" << endl;
		cleanGPU(gpuOriginalImage, gpuModifiedImage);
	}
	//cout << "cudaMemcpu to Cpu Worked" << endl << endl;
	imshow("Display Window", cpuOriginalImage);
}
hipError_t mallocGPU(unsigned char** orig, unsigned char** modif, int width, int height) {
	
	//variables for the mallocing 
	hipError_t gpuStatus = hipSuccess; //test var
	int mallocSize = width * height * sizeof(unsigned char);//size var, for readablity
	cout << "malloc Size: " << mallocSize << endl;//test the size malloced
	//set device and malloc data space
	try{
		gpuStatus = hipSetDevice(0);
		if (gpuStatus != hipSuccess) {
			throw("hipSetDevice failed");
		}
		cout << "hipSetDevice was a success" << endl;

		//malloc space for originalImage.data
			//NOT & Bc that is passing the adress of the thing holding the adress of the pointer, and all else gets lost 
		gpuStatus = hipMalloc((void**)orig, mallocSize); 
		if (gpuStatus != hipSuccess) {
			throw("hipMalloc gpuOriginal failed");
		}
		cout << "hipMalloc gpuOriginal was a success" << endl;
		//malloc space for modifiedImage.data
		gpuStatus = hipMalloc((void**)modif, mallocSize);
		if (gpuStatus != hipSuccess) {
			throw("hipMalloc gpuModified failed");
		}
		cout << "hipMalloc gpuModified was a success" << endl;
	}
	catch (char* errMsg) {
		cout << "Error: " << errMsg << endl << endl;
		cleanGPU(*orig, *modif);
	}

	return gpuStatus;
}
hipError_t copyToGPU(unsigned char* gpuData, unsigned char* cpuData, int width, int height) {
	hipError_t copySucess = hipSuccess;//test var
	int cpySize = width * height * sizeof(unsigned char);//size var, for readability
	cout << "Copy to GPU Size: "<<cpySize << endl;//test size

	//copy the data from the original image onto the malloced space on the gpu
	copySucess = hipMemcpy(gpuData, cpuData, cpySize, hipMemcpyHostToDevice);
	if (copySucess != hipSuccess) {
		cout << "Error: hipMemcpy Cpu to Gpu failed" << endl << endl;
	}

	return copySucess;
}
void cleanGPU(unsigned char* orig, unsigned char* modif) {
	
	hipError_t cleanStatus= hipSuccess;
	
	cleanStatus = hipFree(orig);
	if (cleanStatus == hipSuccess) {
		cout << "Origial Data was freed from the GPU" << endl;
	}
	cleanStatus = hipFree(modif);
	if (cleanStatus == hipSuccess) {
		cout << "Modified Data was freed from the GPU" << endl;
	}
}
hipError_t copyToCPU(unsigned char* gpuData, unsigned char* cpuData, int width, int height) {
	hipError_t copySuccess= hipSuccess;
	int copySize = width * height * sizeof(unsigned char);
	//cout << "Copy To CPU Size: " << copySize << endl;
	
	//copy data of the modified image from the gpu back to the gpu
	copySuccess = hipMemcpy(cpuData, gpuData, copySize, hipMemcpyDeviceToHost);
	if (copySuccess != hipSuccess) {
		cout << "Error: hipMemcpy Gpu to Cpu failed" << endl << endl;
	}

	return copySuccess;
}